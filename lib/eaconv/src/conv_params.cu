#include "eaconv/src/cuda_check.h"
#include "eaconv/src/conv_params.h"

Convolution_Params::Convolution_Params(int stride_x,
                                       int stride_y,
                                       int padding_x,
                                       int padding_y,
                                       int dilation_x,
                                       int dilation_y,
                                       int input_batch_size,
                                       int input_channels,
                                       int input_h,
                                       int input_w,
                                       int kernel_out,
                                       int kernel_in,
                                       int kernel_h,
                                       int kernel_w,
                                       int output_batch_size,
                                       int output_channels,
                                       int output_h,
                                       int output_w) {
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_desc));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_desc,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/input_batch_size,
                                        /*channels=*/input_channels,
                                        /*image_height=*/input_h,
                                        /*image_width=*/input_w));
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_desc));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_desc,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/output_batch_size,
                                        /*channels=*/output_channels,
                                        /*image_height=*/output_h,
                                        /*image_width=*/output_w));
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_desc));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_desc,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*out_channels=*/kernel_out,
                                        /*in_channels=*/kernel_in,
                                        /*kernel_height=*/kernel_h,
                                        /*kernel_width=*/kernel_w));
  checkCUDNN(hipdnnCreateTensorDescriptor(&bias_desc));
  checkCUDNN(hipdnnSetTensor4dDescriptor(bias_desc,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        1,
                                        /*channels=*/output_channels,
                                        1,
                                        1));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    /*pad_height=*/padding_x,
    /*pad_width=*/padding_y,
    /*vertical_stride=*/stride_x,
    /*horizontal_stride=*/stride_y,
    /*dilation_height=*/dilation_x,
    /*dilation_width=*/dilation_y,
    /*mode=*/HIPDNN_CROSS_CORRELATION,
    /*computeType=*/HIPDNN_DATA_FLOAT));
  checkCUDNN(hipdnnSetConvolutionMathType(conv_desc,
                                         HIPDNN_TENSOR_OP_MATH));
}

Convolution_Params::~Convolution_Params() {
  hipdnnDestroyTensorDescriptor(input_desc);
  hipdnnDestroyTensorDescriptor(output_desc);
  hipdnnDestroyFilterDescriptor(kernel_desc);
  hipdnnDestroyTensorDescriptor(bias_desc);
  hipdnnDestroyConvolutionDescriptor(conv_desc);
}
