// Copyright (c) 2016-     Facebook, Inc            (Adam Paszke)
// Copyright (c) 2014-     Facebook, Inc            (Soumith Chintala)
// Copyright (c) 2011-2014 Idiap Research Institute (Ronan Collobert)
// Copyright (c) 2012-2014 Deepmind Technologies    (Koray Kavukcuoglu)
// Copyright (c) 2011-2012 NEC Laboratories America (Koray Kavukcuoglu)
// Copyright (c) 2011-2013 NYU                      (Clement Farabet)
// Copyright (c) 2006-2010 NEC Laboratories America (Ronan Collobert, Leon Bottou, Iain Melvin, Jason Weston) // NOLINT
// Copyright (c) 2006      Idiap Research Institute (Samy Bengio)
// Copyright (c) 2001-2004 Idiap Research Institute (Ronan Collobert, Samy Bengio, Johnny Mariethoz) // NOLINT
//
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//
// 3. Neither the names of Facebook, Deepmind Technologies, NYU, NEC Laboratories America // NOLINT
//    and IDIAP Research Institute nor the names of its contributors may be
//    used to endorse or promote products derived from this software without
//    specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.

#include <unordered_map>
#include <mutex> // NOLINT

#include "eaconv/src/handle.h"
#include "eaconv/src/cuda_check.h"

#ifdef __cplusplus
    extern "C" {
#endif

struct Handle {
  hipdnnHandle_t handle;
  Handle() : handle(NULL) {
    checkCUDNN(hipdnnCreate(&handle));
  }
  ~Handle() {
    if (handle) {
      hipdnnDestroy(handle);
    }
  }
};

std::mutex mutex;
std::unordered_map<int, Handle> handles;

hipdnnHandle_t getCudnnHandle(void) {
  int device;
  checkCUDA(hipGetDevice(&device));

  std::lock_guard<std::mutex> guard(mutex);
  return handles[device].handle;
}

#ifdef __cplusplus
    }
#endif
